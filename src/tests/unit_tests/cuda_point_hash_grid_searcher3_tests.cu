#include "hip/hip_runtime.h"
// Copyright (c) 2017 Doyub Kim
//
// I am making my contributions/submissions to this project solely in my
// personal capacity and am not conveying any rights to any intellectual
// property of any third parties.

#include <gtest/gtest.h>

#include <jet/array1.h>
#include <jet/cuda_array1.h>
#include <jet/cuda_point_hash_grid_searcher3.h>
#include <jet/point_parallel_hash_grid_searcher3.h>

#include <vector>

using namespace jet;
using namespace experimental;

TEST(CudaPointHashGridSearcher3, Build) {
    // CPU baseline
    Array1<Vector3D> points = {Vector3D(0, 1, 3), Vector3D(2, 5, 4),
                               Vector3D(-1, 3, 0)};

    PointParallelHashGridSearcher3 searcher(4, 4, 4, std::sqrt(10));
    searcher.build(points.accessor());

    // GPU
    CudaArray1<float4> pointsD(3);
    pointsD[0] = make_float4(0, 1, 3, 0);
    pointsD[1] = make_float4(2, 5, 4, 0);
    pointsD[2] = make_float4(-1, 3, 0, 0);

    CudaPointHashGridSearcher3 searcherD(4, 4, 4, std::sqrt(10.0f));
    searcherD.build(pointsD.view());

    // Compare
    EXPECT_EQ(searcher.keys().size(), searcherD.keys().size());
    EXPECT_EQ(searcher.startIndexTable().size(),
              searcherD.startIndexTable().size());
    EXPECT_EQ(searcher.endIndexTable().size(),
              searcherD.endIndexTable().size());
    EXPECT_EQ(searcher.sortedIndices().size(),
              searcherD.sortedIndices().size());

    for (size_t i = 0; i < searcher.keys().size(); ++i) {
        size_t valD = searcherD.keys()[i];
        EXPECT_EQ(searcher.keys()[i], valD);
    }

    for (size_t i = 0; i < searcher.startIndexTable().size(); ++i) {
        size_t valD = searcherD.startIndexTable()[i];
        EXPECT_EQ(searcher.startIndexTable()[i], valD);
    }

    for (size_t i = 0; i < searcher.endIndexTable().size(); ++i) {
        size_t valD = searcherD.endIndexTable()[i];
        EXPECT_EQ(searcher.endIndexTable()[i], valD);
    }

    for (size_t i = 0; i < searcher.sortedIndices().size(); ++i) {
        size_t valD = searcherD.sortedIndices()[i];
        EXPECT_EQ(searcher.sortedIndices()[i], valD);
    }
}
