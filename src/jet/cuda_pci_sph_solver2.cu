#include "hip/hip_runtime.h"
// Copyright (c) 2018 Doyub Kim
//
// I am making my contributions/submissions to this project solely in my
// personal capacity and am not conveying any rights to any intellectual
// property of any third parties.

#include <pch.h>

#ifdef JET_USE_CUDA

#include <jet/cuda_pci_sph_solver2.h>
#include <jet/cuda_sph_kernels2.h>

using namespace jet;
using namespace experimental;

namespace {

class InitializeBuffersAndComputeForces {
 public:
    inline InitializeBuffersAndComputeForces(
        float m, float h, float2 gravity, float viscosity,
        uint32_t* neighborStarts, uint32_t* neighborEnds,
        uint32_t* neighborLists, float2* positions, float2* velocities,
        float2* smoothedVelocities, float2* forces, float* densities,
        float* pressures, float2* pressureForces, float* densityErrors,
        float* densitiesPredicted)
        : _mass(m),
          _massSquared(m * m),
          _gravity(gravity),
          _viscosity(viscosity),
          _spikyKernel(h),
          _neighborStarts(neighborStarts),
          _neighborEnds(neighborEnds),
          _neighborLists(neighborLists),
          _positions(positions),
          _velocities(velocities),
          _smoothedVelocities(smoothedVelocities),
          _forces(forces),
          _densities(densities),
          _pressures(pressures),
          _pressureForces(pressureForces),
          _densitiesPredicted(densitiesPredicted),
          _densityErrors(densityErrors) {}

    template <typename Index>
    inline JET_CUDA_DEVICE void operator()(Index i) {
        // Initialize buffers
        _pressures[i] = 0.0f;
        _pressureForces[i] = make_float2(0, 0);
        _densityErrors[i] = 0.0f;
        _densitiesPredicted[i] = _densities[i];

        // Compute forces
        uint32_t ns = _neighborStarts[i];
        uint32_t ne = _neighborEnds[i];
        float2 x_i = _positions[i];
        float2 v_i = _velocities[i];
        float d_i = _densities[i];
        float2 f = _gravity;
        float w_i = _mass / d_i;
        float weightSum = w_i * _spikyKernel(0.0f);
        ;
        float2 smoothedVelocity = w_i * v_i;

        for (uint32_t jj = ns; jj < ne; ++jj) {
            uint32_t j = _neighborLists[jj];

            float2 r = _positions[j] - x_i;
            float dist = length(r);

            if (dist > 0.0f) {
                float2 dir = r / dist;

                float2 v_j = _velocities[j];
                float d_j = _densities[j];

                // Viscosity force
                f += _viscosity * _massSquared * (v_j - v_i) / d_j *
                     _spikyKernel.secondDerivative(dist);

                // Pseudo viscosity
                float w_j = _mass / d_j * _spikyKernel(dist);
                weightSum += w_j;
                smoothedVelocity += w_j * v_j;
            }
        }

        _forces[i] = f;

        smoothedVelocity /= weightSum;
        _smoothedVelocities[i] = smoothedVelocity;
    }

 private:
    float _mass;
    float _massSquared;
    float2 _gravity;
    float _viscosity;
    CudaSphSpikyKernel2 _spikyKernel;
    uint32_t* _neighborStarts;
    uint32_t* _neighborEnds;
    uint32_t* _neighborLists;
    float2* _positions;
    float2* _velocities;
    float2* _smoothedVelocities;
    float2* _forces;
    float* _densities;
    float* _pressures;
    float2* _pressureForces;
    float* _densitiesPredicted;
    float* _densityErrors;
};

#define LOWER_X 0.0f
#define UPPER_X 1.0f
#define LOWER_Y 0.0f
#define UPPER_Y 2.0f
#define BND_R 0.0f

class TimeIntegration {
 public:
    TimeIntegration(float dt, float smoothFactor, float2* positions,
                    float2* velocities, float2* newPositions,
                    float2* newVelocities, float2* smoothedVelocities,
                    float2* forces, float2* pressureForces)
        : _dt(dt),
          _smoothFactor(smoothFactor),
          _positions(positions),
          _velocities(velocities),
          _newPositions(newPositions),
          _newVelocities(newVelocities),
          _smoothedVelocities(smoothedVelocities),
          _forces(forces),
          _pressureForces(pressureForces) {}

    template <typename Index>
    inline JET_CUDA_HOST_DEVICE void operator()(Index i) {
        float2 x = _positions[i];
        float2 v = _velocities[i];
        float2 s = _smoothedVelocities[i];
        float2 f = _forces[i];
        float2 pf = _pressureForces[i];

        v = (1.0f - _smoothFactor) * v + _smoothFactor * s;
        v += _dt * (f + pf);
        x += _dt * v;

        // TODO: Replace with collider
        if (x.x > UPPER_X) {
            x.x = UPPER_X;
            v.x *= BND_R;
        }
        if (x.x < LOWER_X) {
            x.x = LOWER_X;
            v.x *= BND_R;
        }
        if (x.y > UPPER_Y) {
            x.y = UPPER_Y;
            v.y *= BND_R;
        }
        if (x.y < LOWER_Y) {
            x.y = LOWER_Y;
            v.y *= BND_R;
        }

        _newPositions[i] = x;
        _newVelocities[i] = v;
    }

 private:
    float _dt;
    float _smoothFactor;
    float2* _positions;
    float2* _velocities;
    float2* _newPositions;
    float2* _newVelocities;
    float2* _smoothedVelocities;
    float2* _forces;
    float2* _pressureForces;
};

class ComputeDensityError {
 public:
    inline ComputeDensityError(float m, float h, float targetDensity,
                               float delta, float negativePressureScale,
                               uint32_t* neighborStarts, uint32_t* neighborEnds,
                               uint32_t* neighborLists, float2* positions,
                               float* densities, float* pressures,
                               float* densityErrors, float* densitiesPredicted)
        : _mass(m),
          _targetDensity(targetDensity),
          _delta(delta),
          _negativePressureScale(negativePressureScale),
          _neighborStarts(neighborStarts),
          _neighborEnds(neighborEnds),
          _neighborLists(neighborLists),
          _positions(positions),
          _densities(densities),
          _pressures(pressures),
          _densitiesPredicted(densitiesPredicted),
          _densityErrors(densityErrors),
          _stdKernel(h) {}

    template <typename Index>
    inline JET_CUDA_DEVICE void operator()(Index i) {
        uint32_t ns = _neighborStarts[i];
        uint32_t ne = _neighborEnds[i];
        float2 x_i = _positions[i];
        float kernelSum = _stdKernel(0.f);

        for (uint32_t jj = ns; jj < ne; ++jj) {
            uint32_t j = _neighborLists[jj];

            float2 r = _positions[j] - x_i;
            float dist = length(r);

            if (dist > 0.0f) {
                kernelSum += _stdKernel(dist);
            }
        }

        float density = _mass * kernelSum;
        float densityError = (density - _targetDensity);
        float pressure = _delta * densityError;

        if (pressure < 0.0f) {
            pressure *= _negativePressureScale;
            densityError *= _negativePressureScale;
        }

        _pressures[i] += pressure;
        _densitiesPredicted[i] = density;
        _densityErrors[i] = densityError;
    }

 private:
    float _mass;
    float _targetDensity;
    float _delta;
    float _negativePressureScale;
    uint32_t* _neighborStarts;
    uint32_t* _neighborEnds;
    uint32_t* _neighborLists;
    float2* _positions;
    float* _densities;
    float* _pressures;
    float* _densitiesPredicted;
    float* _densityErrors;
    CudaSphStdKernel2 _stdKernel;
};

class ComputePressureForces {
 public:
    inline ComputePressureForces(float m, float h, uint32_t* neighborStarts,
                                 uint32_t* neighborEnds,
                                 uint32_t* neighborLists, float2* positions,
                                 float2* pressureForces, float* densities,
                                 float* pressures)
        : _mass(m),
          _massSquared(m * m),
          _spikyKernel(h),
          _neighborStarts(neighborStarts),
          _neighborEnds(neighborEnds),
          _neighborLists(neighborLists),
          _positions(positions),
          _pressureForces(pressureForces),
          _densities(densities),
          _pressures(pressures) {}

    template <typename Index>
    inline JET_CUDA_HOST_DEVICE void operator()(Index i) {
        uint32_t ns = _neighborStarts[i];
        uint32_t ne = _neighborEnds[i];

        float2 x_i = _positions[i];
        float d_i = _densities[i];
        float p_i = _pressures[i];

        float2 f = make_float2(0, 0);

        for (uint32_t jj = ns; jj < ne; ++jj) {
            uint32_t j = _neighborLists[jj];

            float2 r = _positions[j] - x_i;
            float dist = length(r);

            if (dist > 0.0f) {
                float2 dir = r / dist;

                float d_j = _densities[j];
                float p_j = _pressures[j];

                // Pressure force
                f -= _massSquared * (p_i / (d_i * d_i) + p_j / (d_j * d_j)) *
                     _spikyKernel.gradient(dist, dir);
            }
        }

        _pressureForces[i] = f;
    }

 private:
    float _mass;
    float _massSquared;
    CudaSphSpikyKernel2 _spikyKernel;
    uint32_t* _neighborStarts;
    uint32_t* _neighborEnds;
    uint32_t* _neighborLists;
    float2* _positions;
    float2* _pressureForces;
    float* _densities;
    float* _pressures;
};

}  // namespace

void CudaPciSphSolver2::onAdvanceTimeStep(double timeStepInSeconds) {
    auto sph = sphSystemData();

    float dt = static_cast<float>(timeStepInSeconds);
    float mass = sph->mass();
    float h = sph->kernelRadius();
    size_t n = sph->numberOfParticles();

    auto d = sph->densities();
    auto p = sph->pressures();
    auto x = sph->positions();
    auto v = sph->velocities();
    auto s = smoothedVelocities();
    auto f = forces();

    auto xs = tempPositions();
    auto vs = tempVelocities();
    auto pf = pressureForces();
    auto ds = tempDensities();
    auto de = densityErrors();

    float targetDensity = sph->targetDensity();
    float delta = computeDelta(dt);

    float factor = dt * pseudoViscosityCoefficient();
    factor = clamp(factor, 0.0f, 1.0f);

    // Build neighbor searcher
    sph->buildNeighborSearcher();
    sph->buildNeighborListsAndUpdateDensities();
    auto ns = sph->neighborStarts();
    auto ne = sph->neighborEnds();
    auto nl = sph->neighborLists();

    // Initialize buffers and compute non-pressure forces
    thrust::for_each(
        thrust::counting_iterator<size_t>(0),
        thrust::counting_iterator<size_t>(n),

        InitializeBuffersAndComputeForces(
            mass, h, toFloat2(gravity()), viscosityCoefficient(), ns.data(),
            ne.data(), nl.data(), x.data(), v.data(), s.data(), f.data(),
            d.data(), p.data(), pf.data(), ds.data(), de.data()));

    // Prediction-correction
    // unsigned int maxNumIter = 0;
    // float maxDensityError;
    // float densityErrorRatio = 0.0f;

    for (unsigned int k = 0; k < _maxNumberOfIterations; ++k) {
        // Predict velocity / position and resolve collisions
        thrust::for_each(
            thrust::counting_iterator<size_t>(0),
            thrust::counting_iterator<size_t>(n),

            TimeIntegration(dt, 0.0f, x.data(), v.data(), xs.data(), vs.data(),
                            s.data(), f.data(), pf.data()));

        // Compute pressure from density error
        thrust::for_each(
            thrust::counting_iterator<size_t>(0),
            thrust::counting_iterator<size_t>(n),

            ComputeDensityError(mass, h, targetDensity, delta,
                                negativePressureScale(), ns.data(), ne.data(),
                                nl.data(), xs.data(), d.data(), p.data(),
                                de.data(), ds.data()));

        // Compute pressure gradient force
        thrust::for_each(
            thrust::counting_iterator<size_t>(0),
            thrust::counting_iterator<size_t>(n),

            ComputePressureForces(mass, h, ns.data(), ne.data(), nl.data(),
                                  x.data(), pf.data(), ds.data(), p.data()));

        // Compute max density error
    }

    // Accumulate pressure force and time-integrate
    thrust::for_each(thrust::counting_iterator<size_t>(0),
                     thrust::counting_iterator<size_t>(n),

                     TimeIntegration(dt, factor, x.data(), v.data(), x.data(),
                                     v.data(), s.data(), f.data(), pf.data()));
}

#endif  // JET_USE_CUDA
