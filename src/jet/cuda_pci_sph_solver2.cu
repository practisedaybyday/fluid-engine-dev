#include "hip/hip_runtime.h"
// Copyright (c) 2018 Doyub Kim
//
// I am making my contributions/submissions to this project solely in my
// personal capacity and am not conveying any rights to any intellectual
// property of any third parties.

#include <pch.h>

#ifdef JET_USE_CUDA

#include <jet/cuda_pci_sph_solver2.h>
#include <jet/cuda_sph_kernels2.h>

#include <thrust/extrema.h>

using namespace jet;

namespace {

class InitializeBuffersAndComputeForces {
 public:
    inline InitializeBuffersAndComputeForces(
        float m, float h, float2 gravity, float viscosity,
        const uint32_t* neighborStarts, const uint32_t* neighborEnds,
        const uint32_t* neighborLists, const float2* positions,
        const float2* velocities, float2* smoothedVelocities, float2* forces,
        const float* densities, float* pressures, float2* pressureForces,
        float* densityErrors, float* densitiesPredicted)
        : _mass(m),
          _massSquared(m * m),
          _gravity(gravity),
          _viscosity(viscosity),
          _spikyKernel(h),
          _neighborStarts(neighborStarts),
          _neighborEnds(neighborEnds),
          _neighborLists(neighborLists),
          _positions(positions),
          _velocities(velocities),
          _smoothedVelocities(smoothedVelocities),
          _forces(forces),
          _densities(densities),
          _pressures(pressures),
          _pressureForces(pressureForces),
          _densityErrors(densityErrors),
          _densitiesPredicted(densitiesPredicted) {}

    template <typename Index>
    inline JET_CUDA_DEVICE void operator()(Index i) {
        // Initialize buffers
        _pressures[i] = 0.0f;
        _pressureForces[i] = make_float2(0, 0);
        _densityErrors[i] = 0.0f;
        _densitiesPredicted[i] = _densities[i];

        // Compute forces
        uint32_t ns = _neighborStarts[i];
        uint32_t ne = _neighborEnds[i];
        float2 x_i = _positions[i];
        float2 v_i = _velocities[i];
        float d_i = _densities[i];
        float2 f = _gravity;
        float w_i = _mass / d_i * _spikyKernel(0.0f);
        float weightSum = w_i;
        float2 smoothedVelocity = w_i * v_i;

        for (uint32_t jj = ns; jj < ne; ++jj) {
            uint32_t j = _neighborLists[jj];

            float2 r = _positions[j] - x_i;
            float dist = length(r);

            if (dist > 0.0f) {
                float2 dir = r / dist;

                float2 v_j = _velocities[j];
                float d_j = _densities[j];

                // Viscosity force
                f += _viscosity * _massSquared * (v_j - v_i) / d_j *
                     _spikyKernel.secondDerivative(dist);

                // Pseudo viscosity
                float w_j = _mass / d_j * _spikyKernel(dist);
                weightSum += w_j;
                smoothedVelocity += w_j * v_j;
            }
        }

        _forces[i] = f;

        smoothedVelocity /= weightSum;
        _smoothedVelocities[i] = smoothedVelocity;
    }

 private:
    float _mass;
    float _massSquared;
    float2 _gravity;
    float _viscosity;
    CudaSphSpikyKernel2 _spikyKernel;
    const uint32_t* _neighborStarts;
    const uint32_t* _neighborEnds;
    const uint32_t* _neighborLists;
    const float2* _positions;
    const float2* _velocities;
    float2* _smoothedVelocities;
    float2* _forces;
    const float* _densities;
    float* _pressures;
    float2* _pressureForces;
    float* _densitiesPredicted;
    float* _densityErrors;
};

#define BND_R 0.0f

class TimeIntegration {
 public:
    TimeIntegration(float dt, float m, float smoothFactor, float2 lower,
                    float2 upper, float2* positions, float2* velocities,
                    float2* newPositions, float2* newVelocities,
                    float2* smoothedVelocities, float2* forces,
                    float2* pressureForces)
        : _dt(dt),
          _mass(m),
          _smoothFactor(smoothFactor),
          _lower(lower),
          _upper(upper),
          _positions(positions),
          _velocities(velocities),
          _newPositions(newPositions),
          _newVelocities(newVelocities),
          _smoothedVelocities(smoothedVelocities),
          _forces(forces),
          _pressureForces(pressureForces) {}

    template <typename Index>
    inline JET_CUDA_HOST_DEVICE void operator()(Index i) {
        float2 x = _positions[i];
        float2 v = _velocities[i];
        float2 s = _smoothedVelocities[i];
        float2 f = _forces[i];
        float2 pf = _pressureForces[i];

        v = (1.0f - _smoothFactor) * v + _smoothFactor * s;
        v += _dt * (f + pf) / _mass;
        x += _dt * v;

        // TODO: Add proper collider support
        if (x.x > _upper.x) {
            x.x = _upper.x;
            v.x *= BND_R;
        }
        if (x.x < _lower.x) {
            x.x = _lower.x;
            v.x *= BND_R;
        }
        if (x.y > _upper.y) {
            x.y = _upper.y;
            v.y *= BND_R;
        }
        if (x.y < _lower.y) {
            x.y = _lower.y;
            v.y *= BND_R;
        }

        _newPositions[i] = x;
        _newVelocities[i] = v;
    }

 private:
    float _dt;
    float _mass;
    float _smoothFactor;
    float2 _lower;
    float2 _upper;
    float2* _positions;
    float2* _velocities;
    float2* _newPositions;
    float2* _newVelocities;
    float2* _smoothedVelocities;
    float2* _forces;
    float2* _pressureForces;
};

class ComputeDensityError {
 public:
    inline ComputeDensityError(float m, float h, float targetDensity,
                               float delta, float negativePressureScale,
                               const uint32_t* neighborStarts,
                               const uint32_t* neighborEnds,
                               const uint32_t* neighborLists,
                               const float2* positions, float* pressures,
                               float* densityErrors, float* densitiesPredicted)
        : _mass(m),
          _targetDensity(targetDensity),
          _delta(delta),
          _negativePressureScale(negativePressureScale),
          _neighborStarts(neighborStarts),
          _neighborEnds(neighborEnds),
          _neighborLists(neighborLists),
          _positions(positions),
          _pressures(pressures),
          _densityErrors(densityErrors),
          _densitiesPredicted(densitiesPredicted),
          _stdKernel(h) {}

    template <typename Index>
    inline JET_CUDA_DEVICE void operator()(Index i) {
        uint32_t ns = _neighborStarts[i];
        uint32_t ne = _neighborEnds[i];
        float2 x_i = _positions[i];
        float kernelSum = _stdKernel(0.f);

        for (uint32_t jj = ns; jj < ne; ++jj) {
            uint32_t j = _neighborLists[jj];

            float2 r = _positions[j] - x_i;
            float dist = length(r);

            if (dist > 0.0f) {
                kernelSum += _stdKernel(dist);
            }
        }

        float density = _mass * kernelSum;
        float densityError = (density - _targetDensity);
        float pressure = _delta * densityError;

        if (pressure < 0.0f) {
            pressure *= _negativePressureScale;
            densityError *= _negativePressureScale;
        }

        _pressures[i] += pressure;
        _densitiesPredicted[i] = density;
        _densityErrors[i] = densityError;
    }

 private:
    float _mass;
    float _targetDensity;
    float _delta;
    float _negativePressureScale;
    const uint32_t* _neighborStarts;
    const uint32_t* _neighborEnds;
    const uint32_t* _neighborLists;
    const float2* _positions;
    float* _pressures;
    float* _densitiesPredicted;
    float* _densityErrors;
    CudaSphStdKernel2 _stdKernel;
};

class ComputePressureForces {
 public:
    inline ComputePressureForces(float m, float h,
                                 const uint32_t* neighborStarts,
                                 const uint32_t* neighborEnds,
                                 const uint32_t* neighborLists,
                                 const float2* positions,
                                 float2* pressureForces, const float* densities,
                                 const float* pressures)
        : _mass(m),
          _massSquared(m * m),
          _spikyKernel(h),
          _neighborStarts(neighborStarts),
          _neighborEnds(neighborEnds),
          _neighborLists(neighborLists),
          _positions(positions),
          _pressureForces(pressureForces),
          _densities(densities),
          _pressures(pressures) {}

    template <typename Index>
    inline JET_CUDA_HOST_DEVICE void operator()(Index i) {
        uint32_t ns = _neighborStarts[i];
        uint32_t ne = _neighborEnds[i];

        float2 x_i = _positions[i];
        float d_i = _densities[i];
        float p_i = _pressures[i];

        float2 f = make_float2(0, 0);

        for (uint32_t jj = ns; jj < ne; ++jj) {
            uint32_t j = _neighborLists[jj];

            float2 r = _positions[j] - x_i;
            float dist = length(r);

            if (dist > 0.0f) {
                float2 dir = r / dist;

                float d_j = _densities[j];
                float p_j = _pressures[j];

                // Pressure force
                f -= _massSquared * (p_i / (d_i * d_i) + p_j / (d_j * d_j)) *
                     _spikyKernel.gradient(dist, dir);
            }
        }

        _pressureForces[i] = f;
    }

 private:
    float _mass;
    float _massSquared;
    CudaSphSpikyKernel2 _spikyKernel;
    const uint32_t* _neighborStarts;
    const uint32_t* _neighborEnds;
    const uint32_t* _neighborLists;
    const float2* _positions;
    float2* _pressureForces;
    const float* _densities;
    const float* _pressures;
};

}  // namespace

void CudaPciSphSolver2::onAdvanceTimeStep(double timeStepInSeconds) {
    auto sph = sphSystemData();

    // Build neighbor searcher
    sph->buildNeighborSearcher();
    sph->buildNeighborListsAndUpdateDensities();

    auto d = sph->densities();
    auto p = sph->pressures();
    const float targetDensity = sph->targetDensity();

    size_t n = sph->numberOfParticles();
    float mass = sph->mass();
    float h = sph->kernelRadius();
    auto ns = sph->neighborStarts();
    auto ne = sph->neighborEnds();
    auto nl = sph->neighborLists();
    auto x = sph->positions();
    auto v = sph->velocities();
    auto s = smoothedVelocities();
    auto f = forces();

    float dt = static_cast<float>(timeStepInSeconds);
    float factor = dt * pseudoViscosityCoefficient();
    factor = clamp(factor, 0.0f, 1.0f);

    auto xs = tempPositions();
    auto vs = tempVelocities();
    auto pf = pressureForces();
    auto ds = tempDensities();
    auto de = densityErrors();

    float delta = computeDelta(dt);

    // Initialize buffers and compute non-pressure forces
    thrust::for_each(
        thrust::counting_iterator<size_t>(0),
        thrust::counting_iterator<size_t>(n),

        InitializeBuffersAndComputeForces(
            mass, h, toFloat2(gravity()), viscosityCoefficient(), ns.data(),
            ne.data(), nl.data(), x.data(), v.data(), s.data(), f.data(),
            d.data(), p.data(), pf.data(), de.data(), ds.data()));

    // Prediction-correction
    // unsigned int maxNumIter = 0;
    // float maxDensityError;
    // float densityErrorRatio = 0.0f;
    auto lower = toFloat2(container().lowerCorner);
    auto upper = toFloat2(container().upperCorner);

    for (unsigned int k = 0; k < _maxNumberOfIterations; ++k) {
        // Predict velocity / position and resolve collisions
        thrust::for_each(thrust::counting_iterator<size_t>(0),
                         thrust::counting_iterator<size_t>(n),

                         TimeIntegration(dt, mass, 0.0f, lower, upper, x.data(),
                                         v.data(), xs.data(), vs.data(),
                                         s.data(), f.data(), pf.data()));

        // Compute pressure from density error
        thrust::for_each(thrust::counting_iterator<size_t>(0),
                         thrust::counting_iterator<size_t>(n),

                         ComputeDensityError(mass, h, targetDensity, delta,
                                             negativePressureScale(), ns.data(),
                                             ne.data(), nl.data(), xs.data(),
                                             p.data(), de.data(), ds.data()));

        // Compute pressure gradient force
        thrust::for_each(
            thrust::counting_iterator<size_t>(0),
            thrust::counting_iterator<size_t>(n),

            ComputePressureForces(mass, h, ns.data(), ne.data(), nl.data(),
                                  x.data(), pf.data(), ds.data(), p.data()));

        // Compute max density error
        // float minDensityError = *thrust::min_element(de.begin(), de.end());
        // float maxDensityError = *thrust::max_element(de.begin(), de.end());
        // maxDensityError =
        //     std::max(maxDensityError, std::fabsf(minDensityError));

        // float densityErrorRatio = maxDensityError / targetDensity;
        // maxNumIter = k + 1;

        // if (std::fabs(densityErrorRatio) < _maxDensityErrorRatio) {
        //     break;
        // }
    }

    // Accumulate pressure force and time-integrate
    thrust::for_each(
        thrust::counting_iterator<size_t>(0),
        thrust::counting_iterator<size_t>(n),

        TimeIntegration(dt, mass, factor, lower, upper, x.data(), v.data(),
                        x.data(), v.data(), s.data(), f.data(), pf.data()));
}

#endif  // JET_USE_CUDA
