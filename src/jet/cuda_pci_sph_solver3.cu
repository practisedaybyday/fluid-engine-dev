#include "hip/hip_runtime.h"
// Copyright (c) 2017 Doyub Kim
//
// I am making my contributions/submissions to this project solely in my
// personal capacity and am not conveying any rights to any intellectual
// property of any third parties.

#include <pch.h>

#ifdef JET_USE_CUDA

#include <jet/cuda_pci_sph_solver3.h>
#include <jet/sph_kernels3.h>

using namespace jet;
using namespace experimental;

namespace {

inline JET_CUDA_HOST_DEVICE float stdKernel(float d2, float h2, float h3) {
    if (d2 >= h2) {
        return 0.0f;
    } else {
        float x = 1.0f - d2 / h2;
        return 315.0f / (64.0f * kPiF * h3) * x * x * x;
    }
}

class InitializeBuffersAndComputeForces {
 public:
    inline InitializeBuffersAndComputeForces(
        float m, float h, float4 gravity, float viscosity,
        uint32_t* neighborStarts, uint32_t* neighborEnds,
        uint32_t* neighborLists, float4* positions, float4* velocities,
        float4* smoothedVelocities, float4* forces, float* densities,
        float* pressures, float4* pressureForces, float* densityErrors,
        float* densitiesPredicted)
        : _mass(m),
          _massSquared(m * m),
          _gravity(gravity),
          _viscosity(viscosity),
          _spikyKernel(h),
          _neighborStarts(neighborStarts),
          _neighborEnds(neighborEnds),
          _neighborLists(neighborLists),
          _positions(positions),
          _velocities(velocities),
          _smoothedVelocities(smoothedVelocities),
          _forces(forces),
          _densities(densities),
          _pressures(pressures),
          _pressureForces(pressureForces),
          _densitiesPredicted(densitiesPredicted),
          _densitiErrors(densityErrors) {}

    template <typename Index>
    inline JET_CUDA_DEVICE void operator()(Index i) {
        // Initialize buffers
        _pressures[i] = 0.0f;
        _pressureForces[i] = make_float4(0, 0, 0, 0);
        _densitiErrors[i] = 0.0f;
        _densitiesPredicted[i] = _densities[i];

        // Compute forces
        uint32_t ns = _neighborStarts[i];
        uint32_t ne = _neighborEnds[i];
        float4 x_i = _positions[i];
        float4 v_i = _velocities[i];
        float d_i = _densities[i];
        float4 f = _gravity;
        float w_i = _mass / d_i;
        float weightSum = w_i;
        float4 smoothedVelocity = w_i * v_i;

        for (uint32_t jj = ns; jj < ne; ++jj) {
            uint32_t j = _neighborLists[jj];

            float4 r = _positions[j] - x_i;
            float dist = length(r);

            if (dist > 0.0f) {
                float4 dir = r / dist;

                float4 v_j = _velocities[j];
                float d_j = _densities[j];

                // Viscosity force
                f += _viscosity * _massSquared * (v_j - v_i) / d_j *
                     _spikyKernel.secondDerivative(dist);

                // Pseudo viscosity
                float w_j = _mass / d_j * _spikyKernel(dist);
                weightSum += w_j;
                smoothedVelocity += w_j * v_j;
            }
        }

        _forces[i] = f;

        smoothedVelocity /= weightSum;
        _smoothedVelocities[i] = smoothedVelocity;
    }

 private:
    float _mass;
    float _massSquared;
    float4 _gravity;
    float _viscosity;
    CudaSphSpikyKernel3 _spikyKernel;
    uint32_t* _neighborStarts;
    uint32_t* _neighborEnds;
    uint32_t* _neighborLists;
    float4* _positions;
    float4* _velocities;
    float4* _smoothedVelocities;
    float4* _forces;
    float* _densities;
    float* _pressures;
    float4* _pressureForces;
    float* _densitiesPredicted;
    float* _densitiErrors;
};

#define LOWER_X 0.0f
#define UPPER_X 1.0f
#define LOWER_Y 0.0f
#define UPPER_Y 1.0f
#define LOWER_Z 0.0f
#define UPPER_Z 1.0f
#define BND_R -0.5f

class TimeIntegration {
 public:
    TimeIntegration(float dt, float smoothFactor, float4* positions,
                    float4* velocities, float4* newPositions,
                    float4* newVelocities, float4* smoothedVelocities,
                    float4* forces, float4* pressureForces)
        : _dt(dt),
          _smoothFactor(smoothFactor),
          _positions(positions),
          _velocities(velocities),
          _newPositions(newPositions),
          _newVelocities(newVelocities),
          _smoothedVelocities(smoothedVelocities),
          _forces(forces),
          _pressureForces(pressureForces) {}

    template <typename Index>
    inline JET_CUDA_HOST_DEVICE void operator()(Index i) {
        float4 x = _positions[i];
        float4 v = _velocities[i];
        float4 s = _smoothedVelocities[i];
        float4 f = _forces[i];
        float4 pf = _pressureForces[i];

        v = (1.0f - _smoothFactor) * v + _smoothFactor * s;
        v += _dt * (f + pf);
        x += _dt * v;

        // TODO: Replace with collider
        if (x.x > UPPER_X) {
            x.x = UPPER_X;
            v.x *= BND_R;
        }
        if (x.x < LOWER_X) {
            x.x = LOWER_X;
            v.x *= BND_R;
        }
        if (x.y > UPPER_Y) {
            x.y = UPPER_Y;
            v.y *= BND_R;
        }
        if (x.y < LOWER_Y) {
            x.y = LOWER_Y;
            v.y *= BND_R;
        }
        if (x.z > UPPER_Z) {
            x.z = UPPER_Z;
            v.z *= BND_R;
        }
        if (x.z < LOWER_Z) {
            x.z = LOWER_Z;
            v.z *= BND_R;
        }

        _newPositions[i] = x;
        _newVelocities[i] = v;
    }

 private:
    float _dt;
    float _smoothFactor;
    float4* _positions;
    float4* _velocities;
    float4* _newPositions;
    float4* _newVelocities;
    float4* _smoothedVelocities;
    float4* _forces;
    float4* _pressureForces;
};

class ComputeDensityError {
 public:
    inline ComputeDensityError(float m, float h, uint32_t* neighborStarts,
                               uint32_t* neighborEnds, uint32_t* neighborLists,
                               float4* positions, float* densities,
                               float* pressures, float* densityErrors,
                               float* densitiesPredicted)
        : _mass(m),
          _h2(h * h),
          _h3(h * h * h),
          _neighborStarts(neighborStarts),
          _neighborEnds(neighborEnds),
          _neighborLists(neighborLists),
          _positions(positions),
          _densities(densities),
          _pressures(pressures),
          _densitiesPredicted(densitiesPredicted),
          _densitiErrors(densityErrors) {}

    template <typename Index>
    inline JET_CUDA_DEVICE void operator()(Index i) {
        uint32_t ns = _neighborStarts[i];
        uint32_t ne = _neighborEnds[i];
        float4 x_i = _positions[i];
        float d_i = _densities[i];
        float w_i = _mass / d_i;
        float weightSum = w_i;

        for (uint32_t jj = ns; jj < ne; ++jj) {
            uint32_t j = _neighborLists[jj];

            float4 r = _positions[j] - x_i;
            float dist = length(r);

            if (dist > 0.0f) {
                float4 dir = r / dist;

                float d_j = _densities[j];
                float w_j = _mass / d_j * stdKernel(dist * dist, _h2, _h3);
                weightSum += w_j;
            }
        }

        float density = _mass * weightSum;
        float densityError = (density - _targetDensity);
        float pressure = _delta * densityError;

        if (pressure < 0.0f) {
            pressure *= _negativePressureScale;
            densityError *= _negativePressureScale;
        }

        _pressures[i] += pressure;
        _densitiesPredicted[i] = density;
        _densityErrors[i] = densityError;
    }

 private:
    float _mass;
    float _h2;
    float _h3;
    uint32_t* _neighborStarts;
    uint32_t* _neighborEnds;
    uint32_t* _neighborLists;
    float4* _positions;
    float* _densities;
    float* _pressures;
    float* _densitiesPredicted;
    float* _densitiErrors;
};

class ComputePressureForces {
 public:
    inline ComputePressureForces(float m, float h, uint32_t* neighborStarts,
                                 uint32_t* neighborEnds,
                                 uint32_t* neighborLists, float4* positions,
                                 float4* pressureForces, float* densities,
                                 float* pressures)
        : _mass(m),
          _massSquared(m * m),
          _spikyKernel(h),
          _neighborStarts(neighborStarts),
          _neighborEnds(neighborEnds),
          _neighborLists(neighborLists),
          _positions(positions),
          _pressureForces(forces),
          _densities(densities),
          _pressures(pressures) {}

    template <typename Index>
    inline JET_CUDA_HOST_DEVICE void operator()(Index i) {
        uint32_t ns = _neighborStarts[i];
        uint32_t ne = _neighborEnds[i];

        float4 x_i = _positions[i];
        float d_i = _densities[i];
        float p_i = _pressures[i];

        float w_i = _mass / d_i;
        float weightSum = w_i;

        for (uint32_t jj = ns; jj < ne; ++jj) {
            uint32_t j = _neighborLists[jj];

            float4 r = _positions[j] - x_i;
            float dist = length(r);

            if (dist > 0.0f) {
                float4 dir = r / dist;

                float4 v_j = _velocities[j];
                float d_j = _densities[j];
                float p_j = _pressures[j];

                // Pressure force
                f -= _massSquared * (p_i / (d_i * d_i) + p_j / (d_j * d_j)) *
                     _spikyKernel.gradient(dist, dir);
            }
        }

        _pressureForces[i] = f;
    }

 private:
    float _mass;
    float _massSquared;
    CudaSphSpikyKernel3 _spikyKernel;
    uint32_t* _neighborStarts;
    uint32_t* _neighborEnds;
    uint32_t* _neighborLists;
    float4* _positions;
    float4* _pressureForces;
    float* _densities;
    float* _pressures;
};

}  // namespace

void CudaWcSphSolver3::onAdvanceTimeStep(double timeStepInSeconds) {
    auto sph = sphSystemData();

    float dt = static_cast<float>(timeStepInSeconds);
    float mass = sph->mass();
    float h = sph->kernelRadius();
    size_t n = sph->numberOfParticles();

    auto d = sph->densities();
    auto p = sph->pressures();
    auto ns = sph->neighborStarts();
    auto ne = sph->neighborEnds();
    auto nl = sph->neighborLists();
    auto x = sph->positions();
    auto v = sph->velocities();
    auto s = smoothedVelocities();
    auto f = forces();

    auto xs = tempPositions();
    auto vs = tempVelocities();
    auto pf = pressureForces();
    auto ds = tempDensities();
    auto de = densityErrors();

    // Build neighbor searcher
    sph->buildNeighborSearcher();
    sph->buildNeighborListsAndUpdateDensities();

    // Initialize buffers and compute non-pressure forces
    thrust::for_each(
        thrust::counting_iterator<size_t>(0),
        thrust::counting_iterator<size_t>(n),

        InitializeBuffersAndComputeForces(
            mass, h, toFloat4(gravity(), 0.0f), viscosityCoefficient(),
            ns.data(), ne.data(), nl.data(), x.data(), v.data(), s.data(),
            f.data(), d.data(), p.data(), pf.data(), ds.data(), de.data()));

    // Prediction-correction
    unsigned int maxNumIter = 0;
    float maxDensityError;
    float densityErrorRatio = 0.0f;

    for (unsigned int k = 0; k < _maxNumberOfIterations; ++k) {
        // Predict velocity / position and resolve collisions
        thrust::for_each(
            thrust::counting_iterator<size_t>(0),
            thrust::counting_iterator<size_t>(n),

            TimeIntegration(dt, factor, x.data(), v.data(), xs.data(),
                            vs.data(), s.data(), f.data(), pf.data()));

        // Compute pressure from density error
        thrust::for_each(thrust::counting_iterator<size_t>(0),
                         thrust::counting_iterator<size_t>(n),

                         ComputeDensityError(mass, h, ns.data(), ne.data(),
                                             nl.data(), xs.data(), d.data(),
                                             p.data(), de.data(), ds.data()));

        // Compute pressure gradient force
        thrust::for_each(ComputePressureForces(mass, h, ns.data(), ne.data(),
                                               nl.data(), x.data(), pf.data(),
                                               ds.data(), p.data()));

        // Compute max density error
    }

    // Accumulate pressure force and time-integrate
    thrust::for_each(thrust::counting_iterator<size_t>(0),
                     thrust::counting_iterator<size_t>(n),

                     TimeIntegration(dt, factor, x.data(), v.data(), x.data(),
                                     v.data(), s.data(), f.data(), pf.data()));
}

#endif  // JET_USE_CUDA
