#include "hip/hip_runtime.h"
// Copyright (c) 2018 Doyub Kim
//
// I am making my contributions/submissions to this project solely in my
// personal capacity and am not conveying any rights to any intellectual
// property of any third parties.

#include <pch.h>

#ifdef JET_USE_CUDA

#include <jet/cuda_pci_sph_solver3.h>
#include <jet/cuda_sph_kernels3.h>

#include <thrust/extrema.h>

using namespace jet;
using namespace experimental;

namespace {

class InitializeBuffersAndComputeForces {
 public:
    inline InitializeBuffersAndComputeForces(
        float m, float h, float4 gravity, float viscosity,
        const uint32_t* neighborStarts, const uint32_t* neighborEnds,
        const uint32_t* neighborLists, const float4* positions,
        const float4* velocities, float4* smoothedVelocities, float4* forces,
        const float* densities, float* pressures, float4* pressureForces,
        float* densityErrors, float* densitiesPredicted)
        : _mass(m),
          _massSquared(m * m),
          _gravity(gravity),
          _viscosity(viscosity),
          _spikyKernel(h),
          _neighborStarts(neighborStarts),
          _neighborEnds(neighborEnds),
          _neighborLists(neighborLists),
          _positions(positions),
          _velocities(velocities),
          _smoothedVelocities(smoothedVelocities),
          _forces(forces),
          _densities(densities),
          _pressures(pressures),
          _pressureForces(pressureForces),
          _densityErrors(densityErrors),
          _densitiesPredicted(densitiesPredicted) {}

    template <typename Index>
    inline JET_CUDA_DEVICE void operator()(Index i) {
        // Initialize buffers
        _pressures[i] = 0.0f;
        _pressureForces[i] = make_float4(0, 0, 0, 0);
        _densityErrors[i] = 0.0f;
        _densitiesPredicted[i] = _densities[i];

        // Compute forces
        uint32_t ns = _neighborStarts[i];
        uint32_t ne = _neighborEnds[i];
        float4 x_i = _positions[i];
        float4 v_i = _velocities[i];
        float d_i = _densities[i];
        float4 f = _gravity;
        float w_i = _mass / d_i * _spikyKernel(0.0f);
        float weightSum = w_i;
        float4 smoothedVelocity = w_i * v_i;

        for (uint32_t jj = ns; jj < ne; ++jj) {
            uint32_t j = _neighborLists[jj];

            float4 r = _positions[j] - x_i;
            float dist = length(r);

            if (dist > 0.0f) {
                float4 dir = r / dist;

                float4 v_j = _velocities[j];
                float d_j = _densities[j];

                // Viscosity force
                f += _viscosity * _massSquared * (v_j - v_i) / d_j *
                     _spikyKernel.secondDerivative(dist);

                // Pseudo viscosity
                float w_j = _mass / d_j * _spikyKernel(dist);
                weightSum += w_j;
                smoothedVelocity += w_j * v_j;
            }
        }

        _forces[i] = f;

        smoothedVelocity /= weightSum;
        _smoothedVelocities[i] = smoothedVelocity;
    }

 private:
    float _mass;
    float _massSquared;
    float4 _gravity;
    float _viscosity;
    CudaSphSpikyKernel3 _spikyKernel;
    const uint32_t* _neighborStarts;
    const uint32_t* _neighborEnds;
    const uint32_t* _neighborLists;
    const float4* _positions;
    const float4* _velocities;
    float4* _smoothedVelocities;
    float4* _forces;
    const float* _densities;
    float* _pressures;
    float4* _pressureForces;
    float* _densitiesPredicted;
    float* _densityErrors;
};

#define BND_R 0.0f

class TimeIntegration {
 public:
    TimeIntegration(float dt, float m, float smoothFactor, float3 lower,
                    float3 upper, float4* positions, float4* velocities,
                    float4* newPositions, float4* newVelocities,
                    float4* smoothedVelocities, float4* forces,
                    float4* pressureForces)
        : _dt(dt),
          _mass(m),
          _smoothFactor(smoothFactor),
          _lower(lower),
          _upper(upper),
          _positions(positions),
          _velocities(velocities),
          _newPositions(newPositions),
          _newVelocities(newVelocities),
          _smoothedVelocities(smoothedVelocities),
          _forces(forces),
          _pressureForces(pressureForces) {}

    template <typename Index>
    inline JET_CUDA_HOST_DEVICE void operator()(Index i) {
        float4 x = _positions[i];
        float4 v = _velocities[i];
        float4 s = _smoothedVelocities[i];
        float4 f = _forces[i];
        float4 pf = _pressureForces[i];

        v = (1.0f - _smoothFactor) * v + _smoothFactor * s;
        v += _dt * (f + pf) / _mass;
        x += _dt * v;

        // TODO: Add proper collider support
        if (x.x > _upper.x) {
            x.x = _upper.x;
            v.x *= BND_R;
        }
        if (x.x < _lower.x) {
            x.x = _lower.x;
            v.x *= BND_R;
        }
        if (x.y > _upper.y) {
            x.y = _upper.y;
            v.y *= BND_R;
        }
        if (x.y < _lower.y) {
            x.y = _lower.y;
            v.y *= BND_R;
        }
        if (x.z > _upper.z) {
            x.z = _upper.z;
            v.z *= BND_R;
        }
        if (x.z < _lower.z) {
            x.z = _lower.z;
            v.z *= BND_R;
        }

        _newPositions[i] = x;
        _newVelocities[i] = v;
    }

 private:
    float _dt;
    float _mass;
    float _smoothFactor;
    float3 _lower;
    float3 _upper;
    float4* _positions;
    float4* _velocities;
    float4* _newPositions;
    float4* _newVelocities;
    float4* _smoothedVelocities;
    float4* _forces;
    float4* _pressureForces;
};

class ComputeDensityError {
 public:
    inline ComputeDensityError(float m, float h, float targetDensity,
                               float delta, float negativePressureScale,
                               const uint32_t* neighborStarts,
                               const uint32_t* neighborEnds,
                               const uint32_t* neighborLists,
                               const float4* positions, float* pressures,
                               float* densityErrors, float* densitiesPredicted)
        : _mass(m),
          _targetDensity(targetDensity),
          _delta(delta),
          _negativePressureScale(negativePressureScale),
          _neighborStarts(neighborStarts),
          _neighborEnds(neighborEnds),
          _neighborLists(neighborLists),
          _positions(positions),
          _pressures(pressures),
          _densityErrors(densityErrors),
          _densitiesPredicted(densitiesPredicted),
          _stdKernel(h) {}

    template <typename Index>
    inline JET_CUDA_DEVICE void operator()(Index i) {
        uint32_t ns = _neighborStarts[i];
        uint32_t ne = _neighborEnds[i];
        float4 x_i = _positions[i];
        float kernelSum = _stdKernel(0.f);

        for (uint32_t jj = ns; jj < ne; ++jj) {
            uint32_t j = _neighborLists[jj];

            float4 r = _positions[j] - x_i;
            float dist = length(r);

            if (dist > 0.0f) {
                kernelSum += _stdKernel(dist);
            }
        }

        float density = _mass * kernelSum;
        float densityError = (density - _targetDensity);
        float pressure = _delta * densityError;

        if (pressure < 0.0f) {
            pressure *= _negativePressureScale;
            densityError *= _negativePressureScale;
        }

        _pressures[i] += pressure;
        _densitiesPredicted[i] = density;
        _densityErrors[i] = densityError;
    }

 private:
    float _mass;
    float _targetDensity;
    float _delta;
    float _negativePressureScale;
    const uint32_t* _neighborStarts;
    const uint32_t* _neighborEnds;
    const uint32_t* _neighborLists;
    const float4* _positions;
    float* _pressures;
    float* _densitiesPredicted;
    float* _densityErrors;
    CudaSphStdKernel3 _stdKernel;
};

class ComputePressureForces {
 public:
    inline ComputePressureForces(float m, float h,
                                 const uint32_t* neighborStarts,
                                 const uint32_t* neighborEnds,
                                 const uint32_t* neighborLists,
                                 const float4* positions,
                                 float4* pressureForces, const float* densities,
                                 const float* pressures)
        : _mass(m),
          _massSquared(m * m),
          _spikyKernel(h),
          _neighborStarts(neighborStarts),
          _neighborEnds(neighborEnds),
          _neighborLists(neighborLists),
          _positions(positions),
          _pressureForces(pressureForces),
          _densities(densities),
          _pressures(pressures) {}

    template <typename Index>
    inline JET_CUDA_HOST_DEVICE void operator()(Index i) {
        uint32_t ns = _neighborStarts[i];
        uint32_t ne = _neighborEnds[i];

        float4 x_i = _positions[i];
        float d_i = _densities[i];
        float p_i = _pressures[i];

        float4 f = make_float4(0, 0, 0, 0);

        for (uint32_t jj = ns; jj < ne; ++jj) {
            uint32_t j = _neighborLists[jj];

            float4 r = _positions[j] - x_i;
            float dist = length(r);

            if (dist > 0.0f) {
                float4 dir = r / dist;

                float d_j = _densities[j];
                float p_j = _pressures[j];

                // Pressure force
                f -= _massSquared * (p_i / (d_i * d_i) + p_j / (d_j * d_j)) *
                     _spikyKernel.gradient(dist, dir);
            }
        }

        _pressureForces[i] = f;
    }

 private:
    float _mass;
    float _massSquared;
    CudaSphSpikyKernel3 _spikyKernel;
    const uint32_t* _neighborStarts;
    const uint32_t* _neighborEnds;
    const uint32_t* _neighborLists;
    const float4* _positions;
    float4* _pressureForces;
    const float* _densities;
    const float* _pressures;
};

}  // namespace

void CudaPciSphSolver3::onAdvanceTimeStep(double timeStepInSeconds) {
    auto sph = sphSystemData();

    // Build neighbor searcher
    sph->buildNeighborSearcher();
    sph->buildNeighborListsAndUpdateDensities();

    auto d = sph->densities();
    auto p = sph->pressures();
    const float targetDensity = sph->targetDensity();

    size_t n = sph->numberOfParticles();
    float mass = sph->mass();
    float h = sph->kernelRadius();
    auto ns = sph->neighborStarts();
    auto ne = sph->neighborEnds();
    auto nl = sph->neighborLists();
    auto x = sph->positions();
    auto v = sph->velocities();
    auto s = smoothedVelocities();
    auto f = forces();

    float dt = static_cast<float>(timeStepInSeconds);
    float factor = dt * pseudoViscosityCoefficient();
    factor = clamp(factor, 0.0f, 1.0f);

    auto xs = tempPositions();
    auto vs = tempVelocities();
    auto pf = pressureForces();
    auto ds = tempDensities();
    auto de = densityErrors();

    float delta = computeDelta(dt);

    // Initialize buffers and compute non-pressure forces
    thrust::for_each(
        thrust::counting_iterator<size_t>(0),
        thrust::counting_iterator<size_t>(n),

        InitializeBuffersAndComputeForces(
            mass, h, toFloat4(gravity(), 0.0f), viscosityCoefficient(),
            ns.data(), ne.data(), nl.data(), x.data(), v.data(), s.data(),
            f.data(), d.data(), p.data(), pf.data(), de.data(), ds.data()));

    // Prediction-correction
    // unsigned int maxNumIter = 0;
    // float maxDensityError;
    // float densityErrorRatio = 0.0f;
    auto lower = toFloat3(container().lowerCorner);
    auto upper = toFloat3(container().upperCorner);

    for (unsigned int k = 0; k < _maxNumberOfIterations; ++k) {
        // Predict velocity / position and resolve collisions
        thrust::for_each(thrust::counting_iterator<size_t>(0),
                         thrust::counting_iterator<size_t>(n),

                         TimeIntegration(dt, mass, 0.0f, lower, upper, x.data(),
                                         v.data(), xs.data(), vs.data(),
                                         s.data(), f.data(), pf.data()));

        // Compute pressure from density error
        thrust::for_each(thrust::counting_iterator<size_t>(0),
                         thrust::counting_iterator<size_t>(n),

                         ComputeDensityError(mass, h, targetDensity, delta,
                                             negativePressureScale(), ns.data(),
                                             ne.data(), nl.data(), xs.data(),
                                             p.data(), de.data(), ds.data()));

        // Compute pressure gradient force
        thrust::for_each(
            thrust::counting_iterator<size_t>(0),
            thrust::counting_iterator<size_t>(n),

            ComputePressureForces(mass, h, ns.data(), ne.data(), nl.data(),
                                  x.data(), pf.data(), ds.data(), p.data()));

        // Compute max density error
        // float minDensityError = *thrust::min_element(de.begin(), de.end());
        // float maxDensityError = *thrust::max_element(de.begin(), de.end());
        // maxDensityError =
        //     std::max(maxDensityError, std::fabsf(minDensityError));

        // float densityErrorRatio = maxDensityError / targetDensity;
        // maxNumIter = k + 1;

        // if (std::fabs(densityErrorRatio) < _maxDensityErrorRatio) {
        //     break;
        // }
    }

    // Accumulate pressure force and time-integrate
    thrust::for_each(
        thrust::counting_iterator<size_t>(0),
        thrust::counting_iterator<size_t>(n),

        TimeIntegration(dt, mass, factor, lower, upper, x.data(), v.data(),
                        x.data(), v.data(), s.data(), f.data(), pf.data()));
}

#endif  // JET_USE_CUDA
