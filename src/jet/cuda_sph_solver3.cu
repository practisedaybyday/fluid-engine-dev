#include "hip/hip_runtime.h"
// Copyright (c) 2017 Doyub Kim
//
// I am making my contributions/submissions to this project solely in my
// personal capacity and am not conveying any rights to any intellectual
// property of any third parties.

#include <pch.h>

#include <jet/constants.h>
#include <jet/cuda_sph_solver3.h>
#include <jet/cuda_utils.h>
#include <jet/timer.h>

#include <thrust/fill.h>
#include <thrust/for_each.h>
#include <thrust/tuple.h>

#include <algorithm>

using namespace jet;
using namespace experimental;
using thrust::get;
using thrust::make_tuple;
using thrust::make_zip_iterator;

static double kTimeStepLimitBySpeedFactor = 0.4;
static double kTimeStepLimitByForceFactor = 0.25;

namespace {

struct CudaSphSpikyKernel3 {
    float h;
    float h2;
    float h3;
    float h4;
    float h5;

    inline JET_CUDA_HOST_DEVICE CudaSphSpikyKernel3(float h_)
        : h(h_), h2(h * h), h3(h2 * h), h4(h2 * h2), h5(h3 * h2) {}

    inline JET_CUDA_HOST_DEVICE float operator()(float distance) const {
        if (distance >= h) {
            return 0.0f;
        } else {
            float x = 1.0f - distance / h;
            return 15.0f / (kPiF * h3) * x * x * x;
        }
    }

    inline JET_CUDA_HOST_DEVICE float firstDerivative(float distance) const {
        if (distance >= h) {
            return 0.0f;
        } else {
            float x = 1.0f - distance / h;
            return -45.0f / (kPiF * h4) * x * x;
        }
    }

    inline JET_CUDA_HOST_DEVICE float4 gradient(float4 point) const {
        float dist = length(point);
        if (dist > 0.0f) {
            return gradient(dist, point / dist);
        } else {
            return make_float4(0, 0, 0, 0);
        }
    }

    inline JET_CUDA_HOST_DEVICE float4
    gradient(float distance, float4 directionToCenter) const {
        return -firstDerivative(distance) * directionToCenter;
    }

    inline JET_CUDA_HOST_DEVICE float secondDerivative(float distance) const {
        if (distance >= h) {
            return 0.0f;
        } else {
            float x = 1.0f - distance / h;
            return 90.0f / (kPiF * h5) * x;
        }
    }
};

class ComputePressureFunc {
 public:
    inline ComputePressureFunc(float targetDensity, float eosScale,
                               float eosExponent, float negativePressureScale)
        : _targetDensity(targetDensity),
          _eosScale(eosScale),
          _eosExponent(eosExponent),
          _negativePressureScale(negativePressureScale) {}

    template <typename Float>
    inline JET_CUDA_HOST_DEVICE float operator()(Float d) {
        return computePressureFromEos(d, _targetDensity, _eosScale,
                                      _eosExponent, _negativePressureScale);
    }

    template <typename Float>
    inline JET_CUDA_HOST_DEVICE float computePressureFromEos(
        Float density, float targetDensity, float eosScale, float eosExponent,
        float negativePressureScale) {
        // Equation of state
        // (http://www.ifi.uzh.ch/vmml/publications/pcisph/pcisph.pdf)
        float p = eosScale / eosExponent *
                  (powf((density / targetDensity), eosExponent) - 1.0f);

        // Negative pressure scaling
        if (p < 0) {
            p *= negativePressureScale;
        }

        return p;
    }

 private:
    float _targetDensity;
    float _eosScale;
    float _eosExponent;
    float _negativePressureScale;
};

class ComputeForces {
 public:
    inline ComputeForces(float m, float h, float4 gravity, float viscosity,
                         uint32_t* neighborStarts, uint32_t* neighborEnds,
                         uint32_t* neighborLists, float4* positions,
                         float4* velocities, float4* smoothedVelocities,
                         float4* forces, float* densities, float* pressures)
        : _mass(m),
          _massSquared(m * m),
          _gravity(gravity),
          _viscosity(viscosity),
          _spikyKernel(h),
          _neighborStarts(neighborStarts),
          _neighborEnds(neighborEnds),
          _neighborLists(neighborLists),
          _positions(positions),
          _velocities(velocities),
          _smoothedVelocities(smoothedVelocities),
          _forces(forces),
          _densities(densities),
          _pressures(pressures) {}

    template <typename Index>
    inline JET_CUDA_HOST_DEVICE void operator()(Index i) {
        uint32_t ns = _neighborStarts[i];
        uint32_t ne = _neighborEnds[i];

        float4 x_i = _positions[i];
        float4 v_i = _velocities[i];
        float d_i = _densities[i];
        float p_i = _pressures[i];
        float4 f = _gravity;

        float w_i = _mass / d_i;
        float weightSum = w_i;
        float4 smoothedVelocity = w_i * v_i;

        for (uint32_t jj = ns; jj < ne; ++jj) {
            uint32_t j = _neighborLists[jj];

            float4 r = _positions[j] - x_i;
            float dist = length(r);

            if (dist > 0.0f) {
                float4 dir = r / dist;

                float4 v_j = _velocities[j];
                float d_j = _densities[j];
                float p_j = _pressures[j];

                // Pressure force
                f -= _massSquared * (p_i / (d_i * d_i) + p_j / (d_j * d_j)) *
                     _spikyKernel.gradient(dist, dir);

                // Viscosity force
                f += _viscosity * _massSquared * (v_j - v_i) / d_j *
                     _spikyKernel.secondDerivative(dist);

                // Pseudo viscosity
                float w_j = _mass / d_j * _spikyKernel(dist);
                weightSum += w_j;
                smoothedVelocity += w_j * v_j;
            }
        }

        _forces[i] = f;

        smoothedVelocity /= weightSum;
        _smoothedVelocities[i] = smoothedVelocity;
    }

 private:
    float _mass;
    float _massSquared;
    float4 _gravity;
    float _viscosity;
    CudaSphSpikyKernel3 _spikyKernel;
    uint32_t* _neighborStarts;
    uint32_t* _neighborEnds;
    uint32_t* _neighborLists;
    float4* _positions;
    float4* _velocities;
    float4* _smoothedVelocities;
    float4* _forces;
    float* _densities;
    float* _pressures;
};

#define LOWER_X 0.0f
#define UPPER_X 1.0f
#define LOWER_Y 0.0f
#define UPPER_Y 1.0f
#define LOWER_Z 0.0f
#define UPPER_Z 1.0f
#define BND_R -0.5f

class TimeIntegration {
 public:
    TimeIntegration(float dt, float smoothFactor, float4* positions,
                    float4* velocities, float4* smoothedVelocities,
                    float4* forces)
        : _dt(dt),
          _smoothFactor(smoothFactor),
          _positions(positions),
          _velocities(velocities),
          _smoothedVelocities(smoothedVelocities),
          _forces(forces) {}

    template <typename Index>
    inline JET_CUDA_HOST_DEVICE void operator()(Index i) {
        float4 x = _positions[i];
        float4 v = _velocities[i];
        float4 s = _smoothedVelocities[i];
        float4 f = _forces[i];

        v = (1.0f - _smoothFactor) * v + _smoothFactor * s;
        v += _dt * f;
        x += _dt * v;

        if (x.x > UPPER_X) {
            x.x = UPPER_X;
            v.x *= BND_R;
        }
        if (x.x < LOWER_X) {
            x.x = LOWER_X;
            v.x *= BND_R;
        }
        if (x.y > UPPER_Y) {
            x.y = UPPER_Y;
            v.y *= BND_R;
        }
        if (x.y < LOWER_Y) {
            x.y = LOWER_Y;
            v.y *= BND_R;
        }
        if (x.z > UPPER_Z) {
            x.z = UPPER_Z;
            v.z *= BND_R;
        }
        if (x.z < LOWER_Z) {
            x.z = LOWER_Z;
            v.z *= BND_R;
        }

        _positions[i] = x;
        _velocities[i] = v;
    }

 private:
    float _dt;
    float _smoothFactor;
    float4* _positions;
    float4* _velocities;
    float4* _smoothedVelocities;
    float4* _forces;
};

}  // namespace

CudaSphSolver3::CudaSphSolver3()
    : CudaSphSolver3(static_cast<float>(kWaterDensity), 0.1f, 1.8f) {}

CudaSphSolver3::CudaSphSolver3(float targetDensity, float targetSpacing,
                               float relativeKernelRadius)
    : _targetDensity(targetDensity),
      _targetSpacing(targetSpacing),
      _relativeKernelRadius(relativeKernelRadius) {
    _sphSystemData = std::make_shared<CudaSphSystemData3>();

    _forcesIdx = _sphSystemData->addVectorData();
    _smoothedVelIdx = _sphSystemData->addVectorData();

    setIsUsingFixedSubTimeSteps(false);
}

CudaSphSolver3::~CudaSphSolver3() {}

float CudaSphSolver3::dragCoefficient() const { return _dragCoefficient; }

void CudaSphSolver3::setDragCoefficient(float newDragCoefficient) {
    _dragCoefficient = std::max(newDragCoefficient, 0.0f);
}

float CudaSphSolver3::restitutionCoefficient() const {
    return _restitutionCoefficient;
}

void CudaSphSolver3::setRestitutionCoefficient(
    float newRestitutionCoefficient) {
    _restitutionCoefficient = clamp(newRestitutionCoefficient, 0.0f, 1.0f);
}

const Vector3F& CudaSphSolver3::gravity() const { return _gravity; }

void CudaSphSolver3::setGravity(const Vector3F& newGravity) {
    _gravity = newGravity;
}

float CudaSphSolver3::eosExponent() const { return _eosExponent; }

void CudaSphSolver3::setEosExponent(float newEosExponent) {
    _eosExponent = std::max(newEosExponent, 1.0f);
}

float CudaSphSolver3::negativePressureScale() const {
    return _negativePressureScale;
}

void CudaSphSolver3::setNegativePressureScale(float newNegativePressureScale) {
    _negativePressureScale = clamp(newNegativePressureScale, 0.0f, 1.0f);
}

float CudaSphSolver3::viscosityCoefficient() const {
    return _viscosityCoefficient;
}

void CudaSphSolver3::setViscosityCoefficient(float newViscosityCoefficient) {
    _viscosityCoefficient = std::max(newViscosityCoefficient, 0.0f);
}

float CudaSphSolver3::pseudoViscosityCoefficient() const {
    return _pseudoViscosityCoefficient;
}

void CudaSphSolver3::setPseudoViscosityCoefficient(
    float newPseudoViscosityCoefficient) {
    _pseudoViscosityCoefficient = std::max(newPseudoViscosityCoefficient, 0.0f);
}

float CudaSphSolver3::speedOfSound() const { return _speedOfSound; }

void CudaSphSolver3::setSpeedOfSound(float newSpeedOfSound) {
    _speedOfSound = std::max(newSpeedOfSound, kEpsilonF);
}

float CudaSphSolver3::timeStepLimitScale() const { return _timeStepLimitScale; }

void CudaSphSolver3::setTimeStepLimitScale(float newScale) {
    _timeStepLimitScale = std::max(newScale, 0.0f);
}

CudaSphSystemData3* CudaSphSolver3::particleSystemData() {
    return _sphSystemData.get();
}

const CudaSphSystemData3* CudaSphSolver3::particleSystemData() const {
    return _sphSystemData.get();
}

unsigned int CudaSphSolver3::numberOfSubTimeSteps(
    double timeIntervalInSeconds) const {
    auto particles = particleSystemData();
    size_t numberOfParticles = particles->numberOfParticles();
    // auto f = particles->forces();

    const double kernelRadius = particles->kernelRadius();
    const double mass = particles->mass();

    double maxForceMagnitude = 0.0;

    // for (size_t i = 0; i < numberOfParticles; ++i) {
    //     maxForceMagnitude = std::max(maxForceMagnitude, f[i].length());
    // }
    maxForceMagnitude = kGravity;

    double timeStepLimitBySpeed =
        kTimeStepLimitBySpeedFactor * kernelRadius / _speedOfSound;
    double timeStepLimitByForce =
        kTimeStepLimitByForceFactor *
        std::sqrt(kernelRadius * mass / maxForceMagnitude);

    double desiredTimeStep =
        _timeStepLimitScale *
        std::min(timeStepLimitBySpeed, timeStepLimitByForce);

    return static_cast<unsigned int>(
        std::ceil(timeIntervalInSeconds / desiredTimeStep));
}

void CudaSphSolver3::onInitialize() {
    // When initializing the solver, update the collider and emitter state as
    // well since they also affects the initial condition of the simulation.
    Timer timer;
    updateCollider(0.0f);
    JET_INFO << "Update collider took " << timer.durationInSeconds()
             << " seconds";

    timer.reset();
    updateEmitter(0.0f);
    JET_INFO << "Update emitter took " << timer.durationInSeconds()
             << " seconds";
}

void CudaSphSolver3::onAdvanceTimeStep(double timeStepInSeconds) {
    beginAdvanceTimeStep(timeStepInSeconds);

    // Build neighbor searcher
    _sphSystemData->buildNeighborSearcher();
    _sphSystemData->buildNeighborListsAndUpdateDensities();

    // Compute pressure
    auto d = _sphSystemData->densities();
    auto p = _sphSystemData->pressures();
    const float targetDensity = _sphSystemData->targetDensity();
    const float eosScale = targetDensity * square(_speedOfSound) / _eosExponent;
    thrust::transform(
        d.begin(), d.end(), p.begin(),
        ComputePressureFunc(targetDensity, eosScale, eosExponent(),
                            negativePressureScale()));

    // Compute pressure / viscosity forces and smoothed velocity
    size_t n = _sphSystemData->numberOfParticles();
    float mass = _sphSystemData->mass();
    float h = _sphSystemData->kernelRadius();
    auto ns = _sphSystemData->neighborStarts();
    auto ne = _sphSystemData->neighborEnds();
    auto nl = _sphSystemData->neighborLists();
    auto x = _sphSystemData->positions();
    auto v = _sphSystemData->velocities();
    auto s = _sphSystemData->vectorDataAt(_smoothedVelIdx);
    auto f = _sphSystemData->vectorDataAt(_forcesIdx);

    thrust::for_each(
        thrust::counting_iterator<size_t>(0),
        thrust::counting_iterator<size_t>(n),

        ComputeForces(mass, h, toFloat4(_gravity, 0.0f), viscosityCoefficient(),
                      ns.data(), ne.data(), nl.data(), x.data(), v.data(),
                      s.data(), f.data(), d.data(), p.data()));

    // Time-integration
    float dt = static_cast<float>(timeStepInSeconds);
    float factor = dt * _pseudoViscosityCoefficient;
    factor = clamp(factor, 0.0f, 1.0f);

    thrust::for_each(
        thrust::counting_iterator<size_t>(0),
        thrust::counting_iterator<size_t>(n),

        TimeIntegration(dt, factor, x.data(), v.data(), s.data(), f.data()));

    endAdvanceTimeStep(timeStepInSeconds);
}

void CudaSphSolver3::onBeginAdvanceTimeStep(double timeStepInSeconds) {
    // Update collider and emitter
    Timer timer;
    updateCollider(timeStepInSeconds);
    JET_INFO << "Update collider took " << timer.durationInSeconds()
             << " seconds";

    timer.reset();
    updateEmitter(timeStepInSeconds);
    JET_INFO << "Update emitter took " << timer.durationInSeconds()
             << " seconds";
}

void CudaSphSolver3::onEndAdvanceTimeStep(double timeStepInSeconds) {
    UNUSED_VARIABLE(timeStepInSeconds);
}

void CudaSphSolver3::beginAdvanceTimeStep(double timeStepInSeconds) {
    onBeginAdvanceTimeStep(timeStepInSeconds);
}

void CudaSphSolver3::endAdvanceTimeStep(double timeStepInSeconds) {
    onEndAdvanceTimeStep(timeStepInSeconds);
}

void CudaSphSolver3::updateCollider(double timeStepInSeconds) {
    UNUSED_VARIABLE(timeStepInSeconds);
}

void CudaSphSolver3::updateEmitter(double timeStepInSeconds) {
    UNUSED_VARIABLE(timeStepInSeconds);
}

CudaSphSolver3::Builder CudaSphSolver3::builder() { return Builder(); }

//

CudaSphSolver3::Builder& CudaSphSolver3::Builder::withTargetDensity(
    float targetDensity) {
    _targetDensity = targetDensity;
    return (*this);
}

CudaSphSolver3::Builder& CudaSphSolver3::Builder::withTargetSpacing(
    float targetSpacing) {
    _targetSpacing = targetSpacing;
    return (*this);
}

CudaSphSolver3::Builder& CudaSphSolver3::Builder::withRelativeKernelRadius(
    float relativeKernelRadius) {
    _relativeKernelRadius = relativeKernelRadius;
    return (*this);
}

CudaSphSolver3 CudaSphSolver3::Builder::build() const {
    return CudaSphSolver3(_targetDensity, _targetSpacing,
                          _relativeKernelRadius);
}

CudaSphSolver3Ptr CudaSphSolver3::Builder::makeShared() const {
    return std::shared_ptr<CudaSphSolver3>(
        new CudaSphSolver3(_targetDensity, _targetSpacing,
                           _relativeKernelRadius),
        [](CudaSphSolver3* obj) { delete obj; });
}
