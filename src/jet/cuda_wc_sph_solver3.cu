#include "hip/hip_runtime.h"
// Copyright (c) 2018 Doyub Kim
//
// I am making my contributions/submissions to this project solely in my
// personal capacity and am not conveying any rights to any intellectual
// property of any third parties.

#include <pch.h>

#include <jet/constants.h>
#include <jet/cuda_sph_kernels3.h>
#include <jet/cuda_utils.h>
#include <jet/cuda_wc_sph_solver3.h>
#include <jet/timer.h>

#include <thrust/fill.h>
#include <thrust/for_each.h>
#include <thrust/tuple.h>

#include <algorithm>

using namespace jet;
using thrust::get;
using thrust::make_tuple;
using thrust::make_zip_iterator;

namespace {

class ComputePressureFunc {
 public:
    inline ComputePressureFunc(float targetDensity, float eosScale,
                               float eosExponent, float negativePressureScale)
        : _targetDensity(targetDensity),
          _eosScale(eosScale),
          _eosExponent(eosExponent),
          _negativePressureScale(negativePressureScale) {}

    template <typename Float>
    inline JET_CUDA_HOST_DEVICE float operator()(Float d) {
        return computePressureFromEos(d, _targetDensity, _eosScale,
                                      _eosExponent, _negativePressureScale);
    }

    template <typename Float>
    inline JET_CUDA_HOST_DEVICE float computePressureFromEos(
        Float density, float targetDensity, float eosScale, float eosExponent,
        float negativePressureScale) {
        // Equation of state
        // (http://www.ifi.uzh.ch/vmml/publications/pcisph/pcisph.pdf)
        float p = eosScale / eosExponent *
                  (powf((density / targetDensity), eosExponent) - 1.0f);

        // Negative pressure scaling
        if (p < 0) {
            p *= negativePressureScale;
        }

        return p;
    }

 private:
    float _targetDensity;
    float _eosScale;
    float _eosExponent;
    float _negativePressureScale;
};

class ComputeForces {
 public:
    inline ComputeForces(float m, float h, float4 gravity, float viscosity,
                         const uint32_t* neighborStarts,
                         const uint32_t* neighborEnds,
                         const uint32_t* neighborLists, const float4* positions,
                         const float4* velocities, float4* smoothedVelocities,
                         float4* forces, const float* densities,
                         const float* pressures)
        : _mass(m),
          _massSquared(m * m),
          _gravity(gravity),
          _viscosity(viscosity),
          _spikyKernel(h),
          _neighborStarts(neighborStarts),
          _neighborEnds(neighborEnds),
          _neighborLists(neighborLists),
          _positions(positions),
          _velocities(velocities),
          _smoothedVelocities(smoothedVelocities),
          _forces(forces),
          _densities(densities),
          _pressures(pressures) {}

    template <typename Index>
    inline JET_CUDA_HOST_DEVICE void operator()(Index i) {
        uint32_t ns = _neighborStarts[i];
        uint32_t ne = _neighborEnds[i];

        float4 x_i = _positions[i];
        float4 v_i = _velocities[i];
        float d_i = _densities[i];
        float p_i = _pressures[i];
        float4 f = _gravity;

        float w_i = _mass / d_i * _spikyKernel(0.0f);
        float weightSum = w_i;
        float4 smoothedVelocity = w_i * v_i;

        for (uint32_t jj = ns; jj < ne; ++jj) {
            uint32_t j = _neighborLists[jj];

            float4 r = _positions[j] - x_i;
            float dist = length(r);

            if (dist > 0.0f) {
                float4 dir = r / dist;

                float4 v_j = _velocities[j];
                float d_j = _densities[j];
                float p_j = _pressures[j];

                // Pressure force
                f -= _massSquared * (p_i / (d_i * d_i) + p_j / (d_j * d_j)) *
                     _spikyKernel.gradient(dist, dir);

                // Viscosity force
                f += _viscosity * _massSquared * (v_j - v_i) / d_j *
                     _spikyKernel.secondDerivative(dist);

                // Pseudo viscosity
                float w_j = _mass / d_j * _spikyKernel(dist);
                weightSum += w_j;
                smoothedVelocity += w_j * v_j;
            }
        }

        _forces[i] = f;

        smoothedVelocity /= weightSum;
        _smoothedVelocities[i] = smoothedVelocity;
    }

 private:
    float _mass;
    float _massSquared;
    float4 _gravity;
    float _viscosity;
    CudaSphSpikyKernel3 _spikyKernel;
    const uint32_t* _neighborStarts;
    const uint32_t* _neighborEnds;
    const uint32_t* _neighborLists;
    const float4* _positions;
    const float4* _velocities;
    float4* _smoothedVelocities;
    float4* _forces;
    const float* _densities;
    const float* _pressures;
};

#define BND_R 0.0f

class TimeIntegration {
 public:
    TimeIntegration(float dt, float m, float smoothFactor, float3 lower,
                    float3 upper, float4* positions, float4* velocities,
                    float4* smoothedVelocities, float4* forces)
        : _dt(dt),
          _mass(m),
          _smoothFactor(smoothFactor),
          _lower(lower),
          _upper(upper),
          _positions(positions),
          _velocities(velocities),
          _smoothedVelocities(smoothedVelocities),
          _forces(forces) {}

    template <typename Index>
    inline JET_CUDA_HOST_DEVICE void operator()(Index i) {
        float4 x = _positions[i];
        float4 v = _velocities[i];
        float4 s = _smoothedVelocities[i];
        float4 f = _forces[i];

        v = (1.0f - _smoothFactor) * v + _smoothFactor * s;
        v += _dt * f / _mass;
        x += _dt * v;

        // TODO: Add proper collider support
        if (x.x > _upper.x) {
            x.x = _upper.x;
            v.x *= BND_R;
        }
        if (x.x < _lower.x) {
            x.x = _lower.x;
            v.x *= BND_R;
        }
        if (x.y > _upper.y) {
            x.y = _upper.y;
            v.y *= BND_R;
        }
        if (x.y < _lower.y) {
            x.y = _lower.y;
            v.y *= BND_R;
        }
        if (x.z > _upper.z) {
            x.z = _upper.z;
            v.z *= BND_R;
        }
        if (x.z < _lower.z) {
            x.z = _lower.z;
            v.z *= BND_R;
        }

        _positions[i] = x;
        _velocities[i] = v;
    }

 private:
    float _dt;
    float _mass;
    float _smoothFactor;
    float3 _lower;
    float3 _upper;
    float4* _positions;
    float4* _velocities;
    float4* _smoothedVelocities;
    float4* _forces;
};

}  // namespace

void CudaWcSphSolver3::onAdvanceTimeStep(double timeStepInSeconds) {
    auto sph = sphSystemData();

    // Build neighbor searcher
    sph->buildNeighborSearcher();
    sph->buildNeighborListsAndUpdateDensities();

    // Compute pressure
    auto d = sph->densities();
    auto p = sph->pressures();
    const float targetDensity = sph->targetDensity();
    const float eosScale =
        targetDensity * square(speedOfSound()) / _eosExponent;
    thrust::transform(
        d.begin(), d.end(), p.begin(),
        ComputePressureFunc(targetDensity, eosScale, eosExponent(),
                            negativePressureScale()));

    // Compute pressure / viscosity forces and smoothed velocity
    size_t n = sph->numberOfParticles();
    float mass = sph->mass();
    float h = sph->kernelRadius();
    auto ns = sph->neighborStarts();
    auto ne = sph->neighborEnds();
    auto nl = sph->neighborLists();
    auto x = sph->positions();
    auto v = sph->velocities();
    auto s = smoothedVelocities();
    auto f = forces();

    thrust::for_each(thrust::counting_iterator<size_t>(0),
                     thrust::counting_iterator<size_t>(n),

                     ComputeForces(mass, h, toFloat4(gravity(), 0.0f),
                                   viscosityCoefficient(), ns.data(), ne.data(),
                                   nl.data(), x.data(), v.data(), s.data(),
                                   f.data(), d.data(), p.data()));

    // Time-integration
    float dt = static_cast<float>(timeStepInSeconds);
    float factor = dt * pseudoViscosityCoefficient();
    factor = clamp(factor, 0.0f, 1.0f);
    auto lower = toFloat3(container().lowerCorner);
    auto upper = toFloat3(container().upperCorner);

    thrust::for_each(thrust::counting_iterator<size_t>(0),
                     thrust::counting_iterator<size_t>(n),

                     TimeIntegration(dt, mass, factor, lower, upper, x.data(),
                                     v.data(), s.data(), f.data()));
}
